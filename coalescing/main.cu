#include "hip/hip_runtime.h"
#include <algorithm>
#include <numeric>
#include <random>

#include <argparse/argparse.hpp>

#include "common.hpp"


template <typename T>
__global__ void indirect(T *p, const int *off, const size_t n) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = tid; i < n; i += gridDim.x * blockDim.x) {
    int idx = off[i];
    T f = p[idx];
    f += 1;
    p[idx] = f;
  }

}

int main(int argc, char **argv) {

  argparse::Parser parser;

  int n = 10000;
  int nIters = 5;
  int nWarmup = 5;
  parser.add_positional(n);

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  // generate access patterns
  std::vector<int> cHost(n), uHost(n);
  std::iota(cHost.begin(), cHost.end(), 0);
  std::iota(uHost.begin(), uHost.end(), 0);
  std::shuffle(uHost.begin(), uHost.end(), std::mt19937{std::random_device{}()});


  // allocate device data
  float *fDev;
  double *dDev;
  int *cDev, *uDev;
  CUDA_RUNTIME(hipMalloc(&fDev, n * sizeof(*fDev)));
  CUDA_RUNTIME(hipMalloc(&dDev, n * sizeof(*dDev)));
  CUDA_RUNTIME(hipMalloc(&cDev, n * sizeof(int)));
  CUDA_RUNTIME(hipMalloc(&uDev, n * sizeof(int)));

  // copy indices to device
  CUDA_RUNTIME(hipMemcpy(cDev, cHost.data(), cHost.size() * sizeof(int), hipMemcpyDefault));
  CUDA_RUNTIME(hipMemcpy(uDev, uHost.data(), uHost.size() * sizeof(int), hipMemcpyDefault));

  // GPU kernel launch parameters
  dim3 dimBlock(512,1,1);
  dim3 dimGrid(1,1,1);
  dimGrid.x = (n + dimBlock.x - 1) / dimBlock.x;

  for (int i = 0; i < nIters + nWarmup; ++i) {
    indirect<<<dimGrid, dimBlock>>>(fDev, cDev, n);
    CUDA_RUNTIME(hipDeviceSynchronize());
    indirect<<<dimGrid, dimBlock>>>(fDev, uDev, n);
    CUDA_RUNTIME(hipDeviceSynchronize());
    indirect<<<dimGrid, dimBlock>>>(dDev, cDev, n);
    CUDA_RUNTIME(hipDeviceSynchronize());
    indirect<<<dimGrid, dimBlock>>>(dDev, uDev, n);
    CUDA_RUNTIME(hipDeviceSynchronize());
  }

  CUDA_RUNTIME(hipFree(fDev));
  CUDA_RUNTIME(hipFree(dDev));
  CUDA_RUNTIME(hipFree(cDev));
  CUDA_RUNTIME(hipFree(uDev));

  return 0;
}