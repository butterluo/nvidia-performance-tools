#include "hip/hip_runtime.h"
#include <algorithm>
#include <numeric>
#include <random>

#include <argparse/argparse.hpp>

#include "common.hpp"


__global__ void indirect(float *p, int *off, const size_t n) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = tid; i < n; i += gridDim.x * blockDim.x) {
    int idx = off[i];
    float f = p[idx];
    f += 1;
    p[idx] = f;
  }

}



int main(int argc, char **argv) {

  argparse::Parser parser;

  int n = 10000;
  int nIters = 5;
  int nWarmup = 5;
  parser.add_positional(n);

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  // generate access patterns
  std::vector<int> cHost(n), uHost(n);
  std::iota(cHost.begin(), cHost.end(), 0);
  std::iota(uHost.begin(), uHost.end(), 0);
  std::shuffle(uHost.begin(), uHost.end(), std::mt19937{std::random_device{}()});


  // allocate device data
  float *aDev;
  int *cDev, *uDev;
  CUDA_RUNTIME(hipMalloc(&aDev, n * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev, n * sizeof(int)));
  CUDA_RUNTIME(hipMalloc(&uDev, n * sizeof(int)));

  // copy indices to device
  CUDA_RUNTIME(hipMemcpy(cDev, cHost.data(), cHost.size() * sizeof(int), hipMemcpyDefault));
  CUDA_RUNTIME(hipMemcpy(uDev, uHost.data(), uHost.size() * sizeof(int), hipMemcpyDefault));

  // GPU kernel launch parameters
  dim3 dimBlock(512,1,1);
  dim3 dimGrid(1,1,1);
  dimGrid.x = (n + dimBlock.x - 1) / dimBlock.x;

  for (int i = 0; i < nIters + nWarmup; ++i) {
    indirect<<<dimGrid, dimBlock>>>(aDev, cDev, n);
    CUDA_RUNTIME(hipDeviceSynchronize());
    indirect<<<dimGrid, dimBlock>>>(aDev, uDev, n);
    CUDA_RUNTIME(hipDeviceSynchronize());
  }

  CUDA_RUNTIME(hipFree(aDev));
  CUDA_RUNTIME(hipFree(cDev));
  CUDA_RUNTIME(hipFree(uDev));

  return 0;
}