#include "hip/hip_runtime.h"
#include <algorithm>

#include <nvToolsExt.h>

#include <argparse/argparse.hpp>

#include "common.hpp"

/* NOTE: A and C are column major, B is row major
 */
__global__ void mygemm(float *__restrict__ c, //<! [out] and MxN matrix
                       const float *a,        //<! [in] an MxK matrix
                       const float *b,        //<! [in] an KxN matrix
                       const int M, const int N, const int K) {

#define A(_i, _j) a[(_i) + (_j)*M]
#define B(_i, _j) b[(_i)*N + (_j)]
#define C(_i, _j) c[(_i) + (_j)*M]

  int gidx = blockDim.x * blockIdx.x + threadIdx.x;
  int gidy = blockDim.y * blockIdx.y + threadIdx.y;

  for (int i = gidy; i < M; i += gridDim.y * blockDim.y) {
    for (int j = gidx; j < N; j += gridDim.x * blockDim.x) {
      float acc = 0;
      for (int k = 0; k < K; ++k) {
        acc += A(i, k) * B(k, j);
      }
      C(i, j) = acc;
    }
  }

#undef A
#undef B
#undef C
}

int main(int argc, char **argv) {

  argparse::Parser parser;

  // default matrix sizes:
  // A: 1489 x 1493
  // B: 1493 x 1499
  // C: 1489 x 1499
  int m = 1489;
  int n = 1499;
  int k = 1493;

  int nIters = 5;
  int nWarmup = 5;
  bool check = false;
  parser.add_positional(m);
  parser.add_positional(n);
  parser.add_positional(k);
  parser.add_option(nIters, "--iters");
  parser.add_option(nWarmup, "--warmup");
  parser.add_flag(check, "--check");

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  const int64_t flop = int64_t(m) * int64_t(n) * int64_t(k) * 2;

  // initialize host data
  std::cerr << "generate data\n";
  nvtxRangePush("generate data");
  float *aHost, *bHost, *cHost, *cExpected;
  CUDA_RUNTIME(hipHostAlloc(&aHost, m * k * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&bHost, k * n * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cHost, m * n * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cExpected, m * n * sizeof(float), 0));
  std::generate(aHost, aHost + m * k, random_int);
  std::generate(bHost, bHost + k * n, random_int);
  nvtxRangePop();

  // allocate device data
  float *aDev, *bDev, *cDev;
  CUDA_RUNTIME(hipMalloc(&aDev, m * k * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&bDev, k * n * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev, m * n * sizeof(float)));

  // copy data to device
  std::cerr << "transfer to GPU\n";
  nvtxRangePush("host-to-device");
  CUDA_RUNTIME(
      hipMemcpy(aDev, aHost, m * k * sizeof(float), hipMemcpyDefault));
  CUDA_RUNTIME(
      hipMemcpy(bDev, bHost, k * n * sizeof(float), hipMemcpyDefault));
  nvtxRangePop();

  // create events to time GPU kernel
  hipEvent_t start, stop;
  CUDA_RUNTIME(hipEventCreate(&start));
  CUDA_RUNTIME(hipEventCreate(&stop));

  // GPU kernel launch parameters
  dim3 dimBlock(32, 32);
  dim3 dimGrid;
  dimGrid.x = (n + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (m + dimBlock.y - 1) / dimBlock.y;

  // total elapsed time
  float elapsed = 0;

  /* Launch the kernel nIters + nWarmup times
     Check for correctness on the first time.
     Record the time after nWarmup runs complete.
  */
  for (int i = 0; i < nIters + nWarmup; ++i) {
    nvtxRangePush("kernel");
    CUDA_RUNTIME(hipEventRecord(start));
    mygemm<<<dimGrid, dimBlock>>>(cDev, aDev, bDev, m, n, k);
    CUDA_RUNTIME(hipEventRecord(stop));
    CUDA_RUNTIME(hipEventSynchronize(stop));
    nvtxRangePop();

    // check result once
    if (check && 0 == i) {
      // copy result to host
      CUDA_RUNTIME(
          hipMemcpy(cHost, cDev, m * n * sizeof(float), hipMemcpyDefault));

      // check result on host
      cpu_gemm(cExpected, aHost, bHost, m, n, k);

      for (size_t i = 0; i < m * n; ++i) {
        if (!equal(cExpected[i], cHost[i], 1e-6)) {
          std::cerr << "Error!\n";
          exit(EXIT_FAILURE);
        }
      }
    }

    float millis;
    CUDA_RUNTIME(hipEventElapsedTime(&millis, start, stop));
    std::cerr << i << ": " << millis << (i >= nWarmup ? " *" : " ") << "\n";

    // record time after warmup runs
    if (i >= nWarmup) {
      elapsed += millis;
    }
  }

  // print results
  double gflops = flop / ((elapsed / nIters) / 1000) / 1e9;
  std::cerr << "kernel " << gflops << "GFLOPS (" << flop << " flop, "
            << (elapsed / nIters) / 1000 << "s)\n";

  // release resources
  CUDA_RUNTIME(hipEventDestroy(start));
  CUDA_RUNTIME(hipEventDestroy(stop));
  CUDA_RUNTIME(hipFree(aDev));
  CUDA_RUNTIME(hipFree(bDev));
  CUDA_RUNTIME(hipFree(cDev));
  CUDA_RUNTIME(hipHostFree(aHost));
  CUDA_RUNTIME(hipHostFree(bHost));
  CUDA_RUNTIME(hipHostFree(cHost));
  CUDA_RUNTIME(hipHostFree(cExpected));
  return 0;
}