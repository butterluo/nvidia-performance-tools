#include "hip/hip_runtime.h"
#include <algorithm>

#include <nvToolsExt.h>

#include <argparse/argparse.hpp>

#include "common.hpp"

/* NOTE: A and C are column major, B is row major
 */
__global__ void mygemm(float *__restrict__ c, //<! [out] and MxN matrix
                       const float *a,        //<! [in] an MxK matrix
                       const float *b,        //<! [in] an KxN matrix
                       const int M, const int N, const int K) {

#define A(_i, _j) a[(_i) + (_j)*M]
#define B(_i, _j) b[(_i)*N + (_j)]
#define C(_i, _j) c[(_i) + (_j)*M]

  int gidx = blockDim.x * blockIdx.x + threadIdx.x;
  int gidy = blockDim.y * blockIdx.y + threadIdx.y;

  for (int i = gidy; i < M; i += gridDim.y * blockDim.y) {
    for (int j = gidx; j < N; j += gridDim.x * blockDim.x) {
      float acc = 0;
      for (int k = 0; k < K; ++k) {
        acc += A(i, k) * B(k, j);
      }
      C(i, j) = acc;
    }
  }

#undef A
#undef B
#undef C
}

/* Time the total transfer & matrix-multiplication time
 */
 int main(int argc, char **argv) {

  argparse::Parser parser;

  // default matrix sizes:
  // A: 1600 x 1500
  // B: 1500 x 1400
  // C: 1600 x 1400
  int m = 1600;
  int n = 1400;
  int k = 1500;

  int nIters = 5;
  int nWarmup = 5;
  parser.add_positional(m);
  parser.add_positional(n);
  parser.add_positional(k);
  parser.add_option(nIters, "--iters");
  parser.add_option(nWarmup, "--warmup");

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  const int64_t flop = int64_t(m) * int64_t(n) * int64_t(k) * 2 * nIters;

  // initialize host data
  std::cout << "generate data\n";
  nvtxRangePush("generate data");
  float *aHost, *bHost, *cHost;
  CUDA_RUNTIME(hipHostAlloc(&aHost, m * k * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&bHost, k * n * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cHost, m * n * sizeof(float), 0));
  std::generate(aHost, aHost + m * k, random_int);
  std::generate(bHost, bHost + k * n, random_int);
  nvtxRangePop();

  // allocate device data
  float *aDev, *bDev, *cDev;
  CUDA_RUNTIME(hipMalloc(&aDev, m * k * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&bDev, k * n * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev, m * n * sizeof(float)));

  // create events to time GPU kernel
  hipEvent_t start, stop;
  CUDA_RUNTIME(hipEventCreate(&start));
  CUDA_RUNTIME(hipEventCreate(&stop));

  // GPU kernel launch parameters
  dim3 dimBlock(32, 32);
  dim3 dimGrid;
  dimGrid.x = (n + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (m + dimBlock.y - 1) / dimBlock.y;

  float kernelTime = 0;
  float wallTime = 0;

  for (int iter = 0; iter < nWarmup + nIters; ++iter) {

    auto wallStart = Clock::now();

    // copy data to device
    nvtxRangePush("host-to-device");
    CUDA_RUNTIME(
        hipMemcpy(aDev, aHost, m * k * sizeof(float), hipMemcpyDefault));
    CUDA_RUNTIME(
        hipMemcpy(bDev, bHost, k * n * sizeof(float), hipMemcpyDefault));
    nvtxRangePop();

    // kernel time
    float millis;
    CUDA_RUNTIME(hipEventRecord(start));
    mygemm<<<dimGrid, dimBlock>>>(cDev, aDev, bDev, m, n, k);
    CUDA_RUNTIME(hipEventRecord(stop));
    CUDA_RUNTIME(hipEventSynchronize(stop));
    CUDA_RUNTIME(hipEventElapsedTime(&millis, start, stop));

    // copy data back to host
    nvtxRangePush("device-to-host");
    CUDA_RUNTIME(
        hipMemcpy(cHost, cDev, m * n * sizeof(float), hipMemcpyDefault));
    nvtxRangePop();
    CUDA_RUNTIME(hipDeviceSynchronize());

    Duration wallElapsed = Clock::now() - wallStart;

    std::cout << iter << " kernel=" << millis / 1000
              << " wall=" << wallElapsed.count()
              << (iter >= nWarmup ? " *" : "  ") << "\n";

    // track time if no longer during warmup
    if (iter >= nWarmup) {
      wallTime += wallElapsed.count();
      kernelTime += millis / 1000; // seconds
    }
  }

  // print results
  double kernelGflops = flop / 1e9 / kernelTime;
  std::cout << "kernel " << kernelGflops << "GFLOPS (" << flop << " flop, "
            << kernelTime << "s)\n";
  double wallGflops = flop / 1e9 / wallTime;
  std::cout << "wall " << wallGflops << "GFLOPS (" << flop << " flop, "
            << wallTime << "s)\n";
  // release resources
  CUDA_RUNTIME(hipEventDestroy(start));
  CUDA_RUNTIME(hipEventDestroy(stop));
  CUDA_RUNTIME(hipFree(aDev));
  CUDA_RUNTIME(hipFree(bDev));
  CUDA_RUNTIME(hipFree(cDev));
  CUDA_RUNTIME(hipHostFree(aHost));
  CUDA_RUNTIME(hipHostFree(bHost));
  CUDA_RUNTIME(hipHostFree(cHost));
  return 0;
}