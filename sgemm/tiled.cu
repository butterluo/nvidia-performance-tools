#include "hip/hip_runtime.h"
#include <algorithm>

#include <argparse/argparse.hpp>

#include "common.hpp"

#define TILE_WIDTH 16

/* NOTE: A and C are column major, B is row major
 */
__global__ void mygemm(float *c,       //<! [out] and MxN matrix
                       const float *a, //<! [in] an MxK matrix
                       const float *b, //<! [in] an KxN matrix
                       const int M, const int N, const int K) {

  __shared__ float aSh[TILE_WIDTH][TILE_WIDTH];
  __shared__ float bSh[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = by * TILE_WIDTH + ty;
  int j = bx * TILE_WIDTH + tx;
  float acc = 0;

#define A(_i, _j) a[(_i) + (_j)*M]
#define B(_i, _j) b[(_i)*N + (_j)]
#define C(_i, _j) c[(_i) + (_j)*M]

  for (int m = 0; m < (K - 1) / TILE_WIDTH + 1; ++m) {
    if (i < M && m * TILE_WIDTH + tx < K) {
      aSh[ty][tx] = A(i, m * TILE_WIDTH + tx);
    } else {
      aSh[ty][tx] = 0;
    }
    if (j < N && m * TILE_WIDTH + ty < K) {
      bSh[ty][tx] = B(m * TILE_WIDTH + ty, j);
    } else {
      bSh[ty][tx] = 0;
    }

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      acc += aSh[ty][k] * bSh[k][tx];
    }
    __syncthreads();
  }
  if (i < M && j < N) {
    C(i, j) = acc;
  }

#undef A
#undef B
#undef C
}

int main(int argc, char **argv) {

  argparse::Parser parser;

  // default matrix sizes:
  // A: 307 x 313
  // B: 313 x 311
  // C: 307 x 311
  int m = 307;
  int n = 311;
  int k = 313;

  int nIters = 5;
  int nWarmup = 5;
  bool noCheck = false;
  parser.add_positional(m);
  parser.add_positional(n);
  parser.add_positional(k);
  parser.add_option(nIters, "--iters");
  parser.add_option(nWarmup, "--warmup");
  parser.add_flag(noCheck, "--no-check");

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  const int64_t flop = m * n * k * 2;

  // initialize host data
  std::vector<float> aHost(m * k), bHost(k * n), cHost(m * n), cExpected(m * n);
  std::generate(aHost.begin(), aHost.end(), random_int);
  std::generate(bHost.begin(), bHost.end(), random_int);

  // allocate device data
  float *aDev, *bDev, *cDev;
  CUDA_RUNTIME(hipMalloc(&aDev, aHost.size() * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&bDev, bHost.size() * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev, cHost.size() * sizeof(float)));

  // copy data to device
  CUDA_RUNTIME(hipMemcpy(aDev, aHost.data(), aHost.size() * sizeof(float),
                          hipMemcpyDefault));
  CUDA_RUNTIME(hipMemcpy(bDev, bHost.data(), bHost.size() * sizeof(float),
                          hipMemcpyDefault));

  // create events to time GPU kernel
  hipEvent_t start, stop;
  CUDA_RUNTIME(hipEventCreate(&start));
  CUDA_RUNTIME(hipEventCreate(&stop));

  // GPU kernel launch parameters
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  dim3 dimGrid;
  dimGrid.x = (n + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (m + dimBlock.y - 1) / dimBlock.y;

  // total elapsed time
  float elapsed = 0;

  /* Launch the kernel nIters + nWarmup times
     Check for correctness on the first time.
     Record the time after nWarmup runs complete.
  */
  for (int i = 0; i < nIters + nWarmup; ++i) {
    CUDA_RUNTIME(hipEventRecord(start));
    mygemm<<<dimGrid, dimBlock>>>(cDev, aDev, bDev, m, n, k);
    CUDA_RUNTIME(hipEventRecord(stop));
    CUDA_RUNTIME(hipEventSynchronize(stop));

    // check result once
    if (!noCheck && 0 == i) {
      // copy result to host
      CUDA_RUNTIME(hipMemcpy(cHost.data(), cDev, cHost.size() * sizeof(float),
                              hipMemcpyDefault));

      // check result on host
      cpu_gemm(cExpected.data(), aHost.data(), bHost.data(), m, n, k);

      for (size_t i = 0; i < cExpected.size(); ++i) {
        if (!equal(cExpected[i], cHost[i], 1e-6)) {
          std::cerr << "Error!\n";
          exit(EXIT_FAILURE);
        }
      }
    }

    float millis;
    CUDA_RUNTIME(hipEventElapsedTime(&millis, start, stop));
    std::cerr << i << ": " << millis << (i >= nWarmup ? " *" : " ") << "\n";

    // record time after warmup runs
    if (i >= nWarmup) {
      elapsed += millis;
    }
  }

  // print results
  double gflops = flop / ((elapsed / nIters) / 1000) / 1e9;
  std::cerr << "tiled " << gflops << "GFLOPS (" << flop << " flop, "
            << (elapsed / nIters) / 1000 << "s)\n";

  // release resources
  CUDA_RUNTIME(hipEventDestroy(start));
  CUDA_RUNTIME(hipEventDestroy(stop));
  CUDA_RUNTIME(hipFree(aDev));
  CUDA_RUNTIME(hipFree(bDev));
  CUDA_RUNTIME(hipFree(cDev));
  return 0;
}