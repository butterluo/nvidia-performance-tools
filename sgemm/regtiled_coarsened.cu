#include "hip/hip_runtime.h"
#include <algorithm>

#include <argparse/argparse.hpp>

#include "common.hpp"

#define TILE_SZ_A 128
#define TILE_SZ_B 16
#define TILE_SZ_RATIO (TILE_SZ_A / TILE_SZ_B)

/* NOTE: A and C are column major, B is row major
 */
__global__ void mygemm(float *c,       //<! [out] and MxN matrix
                       const float *a, //<! [in] an MxK matrix
                       const float *b, //<! [in] an KxN matrix
                       const int M, const int N, const int K) {

// Macros for accessing flattened matrices
#define A(_i, _j) a[(_i) + (_j)*M]
#define B(_i, _j) b[(_i)*N + (_j)]
#define C(_i, _j) c[(_i) + (_j)*M]

  // Shared memory for tiling input B array
  __shared__ float B_s[TILE_SZ_RATIO][TILE_SZ_B];

  // Index variables
  const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int col = blockIdx.y * TILE_SZ_B;

  // Privatization of output variables
  float c_reg[TILE_SZ_B];

  // Initialize output values
  for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
    c_reg[outIdx] = 0;
  }

  // Loop over the input tiles
  for (unsigned int tileIdx = 0; tileIdx < (K - 1) / TILE_SZ_RATIO + 1;
       ++tileIdx) {
    // Load the tile of B into shared memory
    const unsigned int i = threadIdx.x / TILE_SZ_B;
    const unsigned int j = threadIdx.x % TILE_SZ_B;
    if (tileIdx * TILE_SZ_RATIO + i < K && col + j < N) {
      B_s[i][j] = B(tileIdx * TILE_SZ_RATIO + i, col + j);
    } else {
      B_s[i][j] = 0;
    }
    __syncthreads();
    // Loop over elements inside the tile
    for (unsigned int idx = 0; idx < TILE_SZ_RATIO; ++idx) {
      // Load tile of A matrix into register
      float a_reg;
      if (row < M && tileIdx * TILE_SZ_RATIO + idx < K) {
        a_reg = A(row, tileIdx * TILE_SZ_RATIO + idx);
      } else {
        a_reg = 0;
      }
      // Loop over and update the output elements assigned to the thread
      for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
        c_reg[outIdx] += a_reg * B_s[idx][outIdx];
      }
    }
    __syncthreads();
  }

  for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
    if (row < M && col + outIdx < N) {
      C(row, col + outIdx) = c_reg[outIdx];
    }
  }

#undef A
#undef B
#undef C
}

int main(int argc, char **argv) {

  argparse::Parser parser;

  // default matrix sizes:
  // A: 307 x 313
  // B: 313 x 311
  // C: 307 x 311
  int m = 307;
  int n = 311;
  int k = 313;

  int nIters = 5;
  int nWarmup = 5;
  parser.add_positional(m);
  parser.add_positional(n);
  parser.add_positional(k);
  parser.add_option(nIters, "--iters");
  parser.add_option(nWarmup, "--warmup");

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  const int64_t flop = m * n * k * 2;

  // initialize host data
  std::vector<float> aHost(m * k), bHost(k * n), cHost(m * n), cExpected(m * n);
  std::generate(aHost.begin(), aHost.end(), random_int);
  std::generate(bHost.begin(), bHost.end(), random_int);

  // allocate device data
  float *aDev, *bDev, *cDev;
  CUDA_RUNTIME(hipMalloc(&aDev, aHost.size() * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&bDev, bHost.size() * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev, cHost.size() * sizeof(float)));

  // copy data to device
  CUDA_RUNTIME(hipMemcpy(aDev, aHost.data(), aHost.size() * sizeof(float),
                          hipMemcpyDefault));
  CUDA_RUNTIME(hipMemcpy(bDev, bHost.data(), bHost.size() * sizeof(float),
                          hipMemcpyDefault));

  // create events to time GPU kernel
  hipEvent_t start, stop;
  CUDA_RUNTIME(hipEventCreate(&start));
  CUDA_RUNTIME(hipEventCreate(&stop));

  // GPU kernel launch parameters
  dim3 dimGrid((m - 1) / TILE_SZ_A + 1, (n - 1) / TILE_SZ_B + 1);
  dim3 dimBlock(TILE_SZ_A, 1);

  // total elapsed time
  float elapsed = 0;

  /* Launch the kernel nIters + nWarmup times
     Check for correctness on the first time.
     Record the time after nWarmup runs complete.
  */
  for (int i = 0; i < nIters + nWarmup; ++i) {
    CUDA_RUNTIME(hipEventRecord(start));
    mygemm<<<dimGrid, dimBlock>>>(cDev, aDev, bDev, m, n, k);
    CUDA_RUNTIME(hipEventRecord(stop));
    CUDA_RUNTIME(hipEventSynchronize(stop));

    // check result once
    if (i == 0) {
      // copy result to host
      CUDA_RUNTIME(hipMemcpy(cHost.data(), cDev, cHost.size() * sizeof(float),
                              hipMemcpyDefault));

      // check result on host
      cpu_gemm(cExpected.data(), aHost.data(), bHost.data(), m, n, k);

      for (size_t i = 0; i < cExpected.size(); ++i) {
        if (!equal(cExpected[i], cHost[i], 1e-6)) {
          std::cerr << "Error!\n";
          exit(EXIT_FAILURE);
        }
      }
    }

    float millis;
    CUDA_RUNTIME(hipEventElapsedTime(&millis, start, stop));
    std::cerr << i << ": " << millis << (i >= nWarmup ? " *" : " ") << "\n";

    // record time after warmup runs
    if (i >= nWarmup) {
      elapsed += millis;
    }
  }

  // print results
  double gflops = flop / ((elapsed / nIters) / 1000) / 1e9;
  std::cerr << gflops << "GFLOPS (" << flop << " flop, "
            << (elapsed / nIters) / 1000 << "s)\n";

  // release resources
  CUDA_RUNTIME(hipEventDestroy(start));
  CUDA_RUNTIME(hipEventDestroy(stop));
  CUDA_RUNTIME(hipFree(aDev));
  CUDA_RUNTIME(hipFree(bDev));
  CUDA_RUNTIME(hipFree(cDev));
  return 0;
}