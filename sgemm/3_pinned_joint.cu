#include "hip/hip_runtime.h"
#include <algorithm>

#include <nvToolsExt.h>

#include <argparse/argparse.hpp>

#include "common.hpp"

#define TILE_SZ_A 64
#define TILE_SZ_B 16
#define TILE_SZ_RATIO (TILE_SZ_A / TILE_SZ_B)

/* NOTE: A and C are column major, B is row major
 */
__global__ void mygemm(float * __restrict__ c,       //<! [out] and MxN matrix
                       const float *a, //<! [in] an MxK matrix
                       const float *b, //<! [in] an KxN matrix
                       const int M, const int N, const int K) {

// Macros for accessing flattened matrices
#define A(_i, _j) a[(_i) + (_j)*M]
#define B(_i, _j) b[(_i)*N + (_j)]
#define C(_i, _j) c[(_i) + (_j)*M]

  // Shared memory for tiling input B array
  __shared__ float B_s[TILE_SZ_RATIO][TILE_SZ_B];

  // Index variables
  const unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int col = blockIdx.y * TILE_SZ_B;

  // Privatization of output variables
  float c_reg[TILE_SZ_B];

  // Initialize output values
  for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
    c_reg[outIdx] = 0;
  }

  // Loop over the input tiles
  for (unsigned int tileIdx = 0; tileIdx < (K - 1) / TILE_SZ_RATIO + 1;
       ++tileIdx) {
    // Load the tile of B into shared memory
    const unsigned int i = threadIdx.x / TILE_SZ_B;
    const unsigned int j = threadIdx.x % TILE_SZ_B;
    if (tileIdx * TILE_SZ_RATIO + i < K && col + j < N) {
      B_s[i][j] = B(tileIdx * TILE_SZ_RATIO + i, col + j);
    } else {
      B_s[i][j] = 0;
    }
    __syncthreads();
    // Loop over elements inside the tile
    for (unsigned int idx = 0; idx < TILE_SZ_RATIO; ++idx) {
      // Load tile of A matrix into register
      float a_reg;
      if (row < M && tileIdx * TILE_SZ_RATIO + idx < K) {
        a_reg = A(row, tileIdx * TILE_SZ_RATIO + idx);
      } else {
        a_reg = 0;
      }
      // Loop over and update the output elements assigned to the thread
      for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
        c_reg[outIdx] += a_reg * B_s[idx][outIdx];
      }
    }
    __syncthreads();
  }

  for (unsigned int outIdx = 0; outIdx < TILE_SZ_B; ++outIdx) {
    if (row < M && col + outIdx < N) {
      C(row, col + outIdx) = c_reg[outIdx];
    }
  }

#undef A
#undef B
#undef C
}

int main(int argc, char **argv) {

  argparse::Parser parser;

  // default matrix sizes:
  // A: 1489 x 1493
  // B: 1493 x 1499
  // C: 1489 x 1499
  int m = 1489;
  int n = 1499;
  int k = 1493;

  int nIters = 5;
  int nWarmup = 5;
  bool check = false;
  parser.add_positional(m);
  parser.add_positional(n);
  parser.add_positional(k);
  parser.add_option(nIters, "--iters");
  parser.add_option(nWarmup, "--warmup");
  parser.add_flag(check, "--check");

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  const int64_t flop = int64_t(m) * int64_t(n) * int64_t(k) * 2;

  // initialize host data
  std::cerr << "generate data\n";
  nvtxRangePush("generate data");
  float *aHost, *bHost, *cHost, *cExpected;
  CUDA_RUNTIME(hipHostAlloc(&aHost, m * k * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&bHost, k * n * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cHost, m * n * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cExpected, m * n * sizeof(float), 0));
  std::generate(aHost, aHost + m * k, random_int);
  std::generate(bHost, bHost + k * n, random_int);
  nvtxRangePop();

  // allocate device data
  float *aDev, *bDev, *cDev;
  CUDA_RUNTIME(hipMalloc(&aDev, m * k * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&bDev, k * n * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev, m * n * sizeof(float)));

  // copy data to device
  std::cerr << "transfer to GPU\n";
  nvtxRangePush("host-to-device");
  CUDA_RUNTIME(
      hipMemcpy(aDev, aHost, m * k * sizeof(float), hipMemcpyDefault));
  CUDA_RUNTIME(
      hipMemcpy(bDev, bHost, k * n * sizeof(float), hipMemcpyDefault));
  nvtxRangePop();

  // create events to time GPU kernel
  hipEvent_t start, stop;
  CUDA_RUNTIME(hipEventCreate(&start));
  CUDA_RUNTIME(hipEventCreate(&stop));

  // GPU kernel launch parameters
  dim3 dimGrid((m + TILE_SZ_A - 1) / TILE_SZ_A, (n +TILE_SZ_B - 1) / TILE_SZ_B);
  dim3 dimBlock(TILE_SZ_A, 1);

  // total elapsed time
  float elapsed = 0;

  /* Launch the kernel nIters + nWarmup times
     Check for correctness on the first time.
     Record the time after nWarmup runs complete.
  */
  for (int i = 0; i < nIters + nWarmup; ++i) {
    CUDA_RUNTIME(hipEventRecord(start));
    mygemm<<<dimGrid, dimBlock>>>(cDev, aDev, bDev, m, n, k);
    CUDA_RUNTIME(hipEventRecord(stop));
    CUDA_RUNTIME(hipEventSynchronize(stop));

    // check result once
    if (check && 0 == i) {
      // copy result to host
      CUDA_RUNTIME(
          hipMemcpy(cHost, cDev, m * n * sizeof(float), hipMemcpyDefault));

      // check result on host
      cpu_gemm(cExpected, aHost, bHost, m, n, k);

      for (size_t i = 0; i < m * n; ++i) {
        if (!equal(cExpected[i], cHost[i], 1e-6)) {
          std::cerr << "Error!\n";
          exit(EXIT_FAILURE);
        }
      }
    }

    float millis;
    CUDA_RUNTIME(hipEventElapsedTime(&millis, start, stop));
    std::cerr << i << ": " << millis << (i >= nWarmup ? " *" : " ") << "\n";

    // record time after warmup runs
    if (i >= nWarmup) {
      elapsed += millis;
    }
  }

  // print results
  double gflops = flop / ((elapsed / nIters) / 1000) / 1e9;
  std::cerr << "kernel " << gflops << "GFLOPS (" << flop << " flop, "
            << (elapsed / nIters) / 1000 << "s)\n";

  // release resources
  CUDA_RUNTIME(hipEventDestroy(start));
  CUDA_RUNTIME(hipEventDestroy(stop));
  CUDA_RUNTIME(hipFree(aDev));
  CUDA_RUNTIME(hipFree(bDev));
  CUDA_RUNTIME(hipFree(cDev));
  CUDA_RUNTIME(hipHostFree(aHost));
  CUDA_RUNTIME(hipHostFree(bHost));
  CUDA_RUNTIME(hipHostFree(cHost));
  CUDA_RUNTIME(hipHostFree(cExpected));
  return 0;
}
