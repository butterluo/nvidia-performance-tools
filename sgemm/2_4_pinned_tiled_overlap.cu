#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>

#include <nvToolsExt.h>

#include <argparse/argparse.hpp>

#include "common.hpp"

#define TILE_WIDTH 32

/* NOTE: A and C are column major, B is row major
 */
__global__ void mygemm(float *__restrict__ c, //<! [out] and MxN matrix
                       const float *a,        //<! [in] an MxK matrix
                       const float *b,        //<! [in] an KxN matrix
                       const int M, const int N, const int K) {

  __shared__ float aSh[TILE_WIDTH][TILE_WIDTH];
  __shared__ float bSh[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = by * TILE_WIDTH + ty;
  int j = bx * TILE_WIDTH + tx;
  float acc = 0;

#define A(_i, _j) a[(_i) + (_j)*M]
#define B(_i, _j) b[(_i)*N + (_j)]
#define C(_i, _j) c[(_i) + (_j)*M]

  for (int m = 0; m < (K - 1) / TILE_WIDTH + 1; ++m) {
    if (i < M && m * TILE_WIDTH + tx < K) {
      aSh[ty][tx] = A(i, m * TILE_WIDTH + tx);
    } else {
      aSh[ty][tx] = 0;
    }
    if (j < N && m * TILE_WIDTH + ty < K) {
      bSh[ty][tx] = B(m * TILE_WIDTH + ty, j);
    } else {
      bSh[ty][tx] = 0;
    }

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      acc += aSh[ty][k] * bSh[k][tx];
    }
    __syncthreads();
  }
  if (i < M && j < N) {
    C(i, j) = acc;
  }

#undef A
#undef B
#undef C
}

int main(int argc, char **argv) {

  argparse::Parser parser;

  // default matrix sizes:
  // A: 1600 x 1500
  // B: 1500 x 1400
  // C: 1600 x 1400
  int m = 1600;
  int n = 1400;
  int k = 1500;

  int nIters = 5;
  int nWarmup = 5;
  parser.add_positional(m);
  parser.add_positional(n);
  parser.add_positional(k);
  parser.add_option(nIters, "--iters");
  parser.add_option(nWarmup, "--warmup");

  if (!parser.parse(argc, argv)) {
    parser.help();
    exit(EXIT_FAILURE);
  }

  // 4 muls of m/2, n/2, k
  const int64_t flop = int64_t(m) / 2 * int64_t(n) / 2 * int64_t(k) * 2 * 4 * nIters;

  // initialize host data
  std::cerr << "generate data\n";
  nvtxRangePush("generate data");
  float *aHost[2], *bHost[2], *cHost[2][2];
  CUDA_RUNTIME(hipHostAlloc(&aHost[0], m / 2 * k * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&aHost[1], m / 2 * k * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&bHost[0], k * n / 2 * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&bHost[1], k * n / 2 * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cHost[0][0], m / 2 * n / 2 * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cHost[0][1], m / 2 * n / 2 * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cHost[1][0], m / 2 * n / 2 * sizeof(float), 0));
  CUDA_RUNTIME(hipHostAlloc(&cHost[1][1], m / 2 * n / 2 * sizeof(float), 0));
  std::generate(aHost[0], aHost[0] + m / 2 * k, random_int);
  std::generate(aHost[1], aHost[1] + m / 2 * k, random_int);
  std::generate(bHost[0], bHost[0] + k * n / 2, random_int);
  std::generate(bHost[1], bHost[1] + k * n / 2, random_int);
  nvtxRangePop();

  // allocate device data
  std::cerr << "allocate data\n";
  float *aDev[2], *bDev[2], *cDev[2][2];
  CUDA_RUNTIME(hipMalloc(&aDev[0], m / 2 * k * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&aDev[1], m / 2 * k * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&bDev[0], k * n / 2 * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&bDev[1], k * n / 2 * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev[0][0], m / 2 * n / 2 * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev[0][1], m / 2 * n / 2 * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev[1][0], m / 2 * n / 2 * sizeof(float)));
  CUDA_RUNTIME(hipMalloc(&cDev[1][1], m / 2 * n / 2 * sizeof(float)));

  // create streams for copy and kernels
  hipStream_t copyStream, kernelStream;
  CUDA_RUNTIME(hipStreamCreate(&copyStream));
  CUDA_RUNTIME(hipStreamCreate(&kernelStream));

  hipEvent_t waitForA0B0, waitForA1, waitForB1, waitC[2][2];
  CUDA_RUNTIME(hipEventCreate(&waitForA0B0));
  CUDA_RUNTIME(hipEventCreate(&waitForA1));
  CUDA_RUNTIME(hipEventCreate(&waitForB1));
  for (int i = 0; i < 2; ++i) {
    for (int j = 0; j < 2; ++j) {
      CUDA_RUNTIME(hipEventCreate(&waitC[i][j]));
    }
  }

  // GPU kernel launch parameters
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  dim3 dimGrid;
  dimGrid.x = (n/2 + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (m/2 + dimBlock.y - 1) / dimBlock.y;

  float kernelTime = 0;
  float wallTime = 0;
  for (int iter = 0; iter < nIters + nWarmup; ++iter) {

    nvtxRangePush("wall time");
    auto wallStart = Clock::now();

    // copy a0 and b0
    CUDA_RUNTIME(hipMemcpyAsync(aDev[0], aHost[0], m / 2 * k * sizeof(float),
                                 hipMemcpyDefault, copyStream));
    CUDA_RUNTIME(hipMemcpyAsync(bDev[0], bHost[0], k * n / 2 * sizeof(float),
                                 hipMemcpyDefault, copyStream));
    CUDA_RUNTIME(hipEventRecord(waitForA0B0, copyStream));

    // have the kernelStream wait for the transfers to complete
    CUDA_RUNTIME(hipStreamWaitEvent(kernelStream, waitForA0B0, 0));

    // launch c[0][0] = a[0] * b[0]
    mygemm<<<dimGrid, dimBlock, 0, kernelStream>>>(cDev[0][0], aDev[0], bDev[0],
                                                   m / 2, n / 2, k);
    CUDA_RUNTIME(hipEventRecord(waitC[0][0], kernelStream));

    // copy a1
    CUDA_RUNTIME(hipMemcpyAsync(aDev[1], aHost[1], m / 2 * k * sizeof(float),
                                 hipMemcpyDefault, copyStream));
    CUDA_RUNTIME(hipEventRecord(waitForA1, kernelStream));

    // launch c[1][0] = a[1] * b[0] after a[1] is on the GPU
    CUDA_RUNTIME(hipStreamWaitEvent(kernelStream, waitForA1, 0));
    mygemm<<<dimGrid, dimBlock, 0, kernelStream>>>(cDev[1][0], aDev[1], bDev[0],
                                                   m / 2, n / 2, k);
    CUDA_RUNTIME(hipEventRecord(waitC[1][0], kernelStream));

    // copy b1
    CUDA_RUNTIME(hipMemcpyAsync(bDev[1], bHost[1], k * n / 2 * sizeof(float),
                                 hipMemcpyDefault, copyStream));
    CUDA_RUNTIME(hipEventRecord(waitForB1, kernelStream));

    // launch c[0][1] = a[0] * b[1] after B1 is on the GPU
    CUDA_RUNTIME(hipStreamWaitEvent(kernelStream, waitForB1, 0));
    mygemm<<<dimGrid, dimBlock, 0, kernelStream>>>(cDev[0][1], aDev[0], bDev[1],
                                                   m / 2, n / 2, k);
    CUDA_RUNTIME(hipEventRecord(waitC[0][1], kernelStream));

    // launch c[1][1] = a[1] * b[1]
    mygemm<<<dimGrid, dimBlock, 0, kernelStream>>>(cDev[1][1], aDev[1], bDev[1],
                                                   m / 2, n / 2, k);
    CUDA_RUNTIME(hipEventRecord(waitC[1][1], kernelStream));

    // copy c back to CPU as kernels finish
    CUDA_RUNTIME(hipStreamWaitEvent(copyStream, waitC[0][0], 0));
    CUDA_RUNTIME(hipMemcpyAsync(cHost[0][0], cDev[0][0],
                                 m / 2 * n / 2 * sizeof(float),
                                 hipMemcpyDefault, copyStream));
    CUDA_RUNTIME(hipStreamWaitEvent(copyStream, waitC[1][0], 0));
    CUDA_RUNTIME(hipMemcpyAsync(cHost[1][0], cDev[1][0],
                                 m / 2 * n / 2 * sizeof(float),
                                 hipMemcpyDefault, copyStream));
    CUDA_RUNTIME(hipStreamWaitEvent(copyStream, waitC[0][1], 0));
    CUDA_RUNTIME(hipMemcpyAsync(cHost[0][1], cDev[0][1],
                                 m / 2 * n / 2 * sizeof(float),
                                 hipMemcpyDefault, copyStream));
    CUDA_RUNTIME(hipStreamWaitEvent(copyStream, waitC[1][1], 0));
    CUDA_RUNTIME(hipMemcpyAsync(cHost[1][1], cDev[1][1],
                                 m / 2 * n / 2 * sizeof(float),
                                 hipMemcpyDefault, copyStream));

    CUDA_RUNTIME(hipDeviceSynchronize());
    nvtxRangePop(); // wall time
    Duration wallElapsed = Clock::now() - wallStart;

    // kernel time
    float kernelElapsed;
    CUDA_RUNTIME(hipEventSynchronize(waitC[1][1]));
    CUDA_RUNTIME(hipEventElapsedTime(&kernelElapsed, waitForA0B0, waitC[1][1]));
    kernelElapsed /= 1000; // seconds

    std::cerr << iter << " kernel=" << kernelElapsed
              << " wall=" << wallElapsed.count()
              << (iter >= nWarmup ? " *" : "  ") << "\n";

    if (iter >= nWarmup) {
      wallTime += wallElapsed.count();
      kernelTime += kernelElapsed;
    }
  }

  // print results
  double kernelGflops = flop / 1e9 / kernelTime;
  std::cerr << "kernel " << kernelGflops << "GFLOPS (" << flop << " flop, "
            << kernelTime << "s)\n";
  double wallGflops = flop / 1e9 / wallTime;
  std::cerr << "wall " << wallGflops << "GFLOPS (" << flop << " flop, "
            << wallTime << "s)\n";
  // release resources

  CUDA_RUNTIME(hipFree(aDev[0]));
  CUDA_RUNTIME(hipFree(aDev[1]));
  CUDA_RUNTIME(hipFree(bDev[0]));
  CUDA_RUNTIME(hipFree(bDev[1]));
  return 0;
}
