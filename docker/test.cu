
#include <hip/hip_runtime.h>
__global__ void kernel(float *a, float *b, int n) {
    *a = *b;
}

int main(void) {
    float *a, *b;
    hipMalloc(&a, 10 * sizeof(float));
    hipMalloc(&b, 10 * sizeof(float));
    kernel<<<1,1>>>(a,b,10);
    hipDeviceSynchronize();
}